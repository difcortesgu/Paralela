#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>
#include <cuda/std/cmath>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/opencv.hpp>
#include <string>
#include <stdio.h>
#include <sys/time.h>

using namespace std;


hipError_t mosaicImage(
    cv::Mat image,
    cv::Mat index_image,
    cv::Mat mosaic_image,
    cv::Mat result_image,
    int tile_size,
    int new_tile_size,
    int n_vertical_tiles,
    int n_horizontal_tiles,
    int n_blocks,
    int n_threads
);

__global__ void mosaicImagekernel(
    const uchar* image,
    int image_width,
    int image_height,
    int image_channels,
    const uchar* index_image,
    int index_image_size,
    const uchar* mosaic_image,
    int mosaic_image_width,
    uchar* result_image,
    int result_image_width,
    int tile_size,
    int new_tile_size,
    int colors_per_thread,
    int blocks,
    int threads,
    int tiles_per_block,
    int tiles_per_thread,
    int tiles_per_row,
    int total_tiles
)
{
    extern __shared__ uchar index[];

    int begin = min(threadIdx.x * colors_per_thread, index_image_size);
    int end = min((threadIdx.x + 1) * colors_per_thread, index_image_size);

    for (int i = begin; i < end; i++)
    {
        index[i * image_channels] = index_image[i * image_channels];
        index[i * image_channels + 1] = index_image[i * image_channels + 1];
        index[i * image_channels + 2] = index_image[i * image_channels + 2];
    }

    __syncthreads();

    // Get iteration boundaries
    int initial_tile = min(min(blockIdx.x * tiles_per_block + threadIdx.x * tiles_per_thread, (blockIdx.x + 1) * tiles_per_block), total_tiles);
    int final_tile = min(min(initial_tile + tiles_per_thread, (blockIdx.x + 1) * tiles_per_block), total_tiles);

    int newPixelB, newPixelG, newPixelR;

    // Loop tiles
    for (int ti = initial_tile; ti < final_tile; ti++)
    {
        // Start of the tile
        int row = (ti / tiles_per_row);
        int col = (ti % tiles_per_row);

        int pos = ( row * image_width * tile_size * image_channels) + (col * tile_size * image_channels);

        newPixelB = 0;
        newPixelG = 0;
        newPixelR = 0;

        // Loop rows of pixels of the tile
        for (int pi = 0; pi < tile_size; pi++)
        {
            // Loop cols of pixels of the tile
            for (int pj = 0; pj < tile_size; pj++)
            {
                // Get pixel
                int pix_pos = pos + (pi * image_width * image_channels) + (pj * image_channels);

                // Add the values to the new pixel
                newPixelB += (int)image[pix_pos];
                newPixelG += (int)image[pix_pos + 1];
                newPixelR += (int)image[pix_pos + 2];
            }
        }

        newPixelB /= (tile_size * tile_size);
        newPixelG /= (tile_size * tile_size);
        newPixelR /= (tile_size * tile_size);

        // Get the closest color
        float minDiff = 10000;
        int closest_pos = 0;

        for (int i = 0; i < index_image_size; i++)
        {
            float diff = norm3df(newPixelB - index[i * image_channels], newPixelG - index[i * image_channels + 1], newPixelR - index[i * image_channels + 2]);
            if (diff < minDiff)
            {
                minDiff = diff;
                closest_pos = i;
            }
        }

        pos = (row * result_image_width * new_tile_size * image_channels) + (col * new_tile_size * image_channels);

        // Update new image pixels with the "pixel" image
        for (int ni = 0; ni < new_tile_size; ni++)
        {
            for (int nj = 0; nj < new_tile_size; nj++)
            {
                int pix_pos = pos + (ni * result_image_width * image_channels) + (nj * image_channels);

                //result_image[pix_pos] = newPixelB;
                //result_image[pix_pos + 1] = newPixelG;
                //result_image[pix_pos + 2] = newPixelR;

                //result_image[pix_pos] = index[closest_pos * image_channels];
                //result_image[pix_pos + 1] = index[closest_pos * image_channels + 1];
                //result_image[pix_pos + 2] = index[closest_pos * image_channels + 2];

                result_image[pix_pos] = mosaic_image[(closest_pos * mosaic_image_width * mosaic_image_width * image_channels) + (ni * mosaic_image_width * image_channels) + (nj * image_channels)];
                result_image[pix_pos + 1] = mosaic_image[(closest_pos * mosaic_image_width * mosaic_image_width * image_channels) + (ni * mosaic_image_width * image_channels) + (nj * image_channels) + 1];
                result_image[pix_pos + 2] = mosaic_image[(closest_pos * mosaic_image_width * mosaic_image_width * image_channels) + (ni * mosaic_image_width * image_channels) + (nj * image_channels) + 2];
            }
        }
        
    }
}

int main(int argc, char** argv)
{
    //Check number of arguments
    if (argc < 5)
    {
        std::cout << "Ingrese todos los argumentos necesarios para ejecutar el proceso" << std::endl;
        return -1;
    }

    int tile_size = 10, new_tile_size = 10, n_blocks = 24, n_threads = 128;

    //Get the arguments
    std::string path_image = argv[1];
    std::string path_save = argv[2];
    std::string path_index_image = "C:\\imagenes\\indexImage.jpg";
    std::string path_mosaic_image = "C:\\imagenes\\mosaicImage.jpg";

    n_blocks = atoi(argv[3]);
    if (n_blocks == 0)
    {
        std::cout << "El número de bloques es invalido" << std::endl;
        return -1;
    }

    n_threads = atoi(argv[4]);
    if (n_threads == 0)
    {
        std::cout << "El número de hilos es invalido" << std::endl;
        return -1;
    }

    tile_size = atoi(argv[5]);
    if (tile_size == 0)
    {
        cout << "El argumento de tamaño es invalido" << endl;
        return -1;
    }

    new_tile_size = atoi(argv[6]);
    if (new_tile_size == 0)
    {
        cout << "El argumento de tamaño nuevo es invalido" << endl;
        return -1;
    }

    cv::Mat image = cv::imread(path_image, cv::IMREAD_COLOR); // Read the file
    if (!image.data)
    {
        std::cout << "Could not open or find the image" << std::endl;
        return -1;
    }

    cv::Mat index_image = cv::imread(path_index_image, cv::IMREAD_COLOR); // Read the file
    if (!index_image.data)
    {
        std::cout << "Could not open or find the index image" << std::endl;
        return -1;
    }

    cv::Mat mosaic_image = cv::imread(path_mosaic_image, cv::IMREAD_COLOR); // Read the file
    if (!mosaic_image.data)
    {
        std::cout << "Could not open or find the mosaic image" << std::endl;
        return -1;
    }

    int h_pad = image.cols % tile_size;
    int v_pad = image.rows % tile_size;

    if (h_pad != 0)
    {
        h_pad = tile_size - h_pad;
    }

    if (v_pad != 0)
    {
        v_pad = tile_size - v_pad;
    }

    //Add padding to the image so that each tile is the same size
    cv::copyMakeBorder(image, image, 0, v_pad, 0, h_pad, cv::BORDER_REFLECT);

    int n_vertical_tiles = image.rows / tile_size;
    int n_horizontal_tiles = image.cols / tile_size;

    cv::Mat result_image = cv::Mat(n_vertical_tiles * new_tile_size, n_horizontal_tiles * new_tile_size, CV_8UC3, cv::Scalar(0, 0, 0));

    // Add vectors in parallel.
    hipError_t cudaStatus = mosaicImage(image, index_image, mosaic_image, result_image, tile_size, new_tile_size, n_vertical_tiles, n_horizontal_tiles, n_blocks, n_threads);
    if (cudaStatus != hipSuccess) {
        printf("%s\n", hipGetErrorString(cudaStatus));
        return 1;
    }

    if (!cv::imwrite(path_save, result_image)) {
        std::cout << "Could not save the image" << std::endl;
        return -1;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        printf("hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

hipError_t mosaicImage(
    cv::Mat image,
    cv::Mat index_image,
    cv::Mat mosaic_image,
    cv::Mat result_image,
    int tile_size,
    int new_tile_size,
    int n_vertical_tiles,
    int n_horizontal_tiles,
    int n_blocks,
    int n_threads
)
{
    //Declare pointers and variables
    uchar* d_image, * d_index_image, * d_mosaic_image, * d_result_image;

    int image_channels = image.channels();

    hipError_t cudaStatus = hipSuccess;

    try
    {
        // Choose which GPU to run on, change this on a multi-GPU system.
        cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess) throw "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?";

        // Allocate GPU buffers.
        cudaStatus = hipMalloc((void**)&d_image, image.rows * image.cols * image.channels() * sizeof(uchar));
        if (cudaStatus != hipSuccess) throw "hipMalloc failed! image";

        cudaStatus = hipMalloc((void**)&d_index_image, index_image.rows * index_image.cols * index_image.channels() * sizeof(uchar));
        if (cudaStatus != hipSuccess) throw "hipMalloc failed! image";

        cudaStatus = hipMalloc((void**)&d_mosaic_image, mosaic_image.rows * mosaic_image.cols * mosaic_image.channels() * sizeof(uchar));
        if (cudaStatus != hipSuccess) throw "hipMalloc failed! image";

        cudaStatus = hipMalloc((void**)&d_result_image, result_image.rows * result_image.cols * result_image.channels() * sizeof(uchar));
        if (cudaStatus != hipSuccess) throw "hipMalloc failed! result image";


        // Copy input vectors from host memory to GPU buffers.
        cudaStatus = hipMemcpy(d_image, image.data, image.rows * image.cols * image.channels() * sizeof(uchar), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) throw "hipMemcpy failed! image to device";

        cudaStatus = hipMemcpy(d_index_image, index_image.data, index_image.rows * index_image.cols * index_image.channels() * sizeof(uchar), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) throw "hipMemcpy failed! result image to device";

        cudaStatus = hipMemcpy(d_mosaic_image, mosaic_image.data, mosaic_image.rows * mosaic_image.cols * mosaic_image.channels() * sizeof(uchar), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) throw "hipMemcpy failed! result image to device";

        int tiles_per_block = std::ceil((float)(n_vertical_tiles * n_horizontal_tiles) / (float)n_blocks);
        int tiles_per_thread = std::ceil((float)tiles_per_block / (float)n_threads);
        int index_image_size = index_image.rows * index_image.cols * index_image.channels() * sizeof(uchar);
        int colors_per_thread = std::ceil((float)index_image.rows / (float)n_threads);

        // Get initial time
        struct timeval tval_before, tval_after, tval_result;
        gettimeofday(&tval_before, NULL);

        // Launch a kernel on the GPU with one thread for each element.
        mosaicImagekernel << <n_blocks, n_threads, index_image_size >> > (d_image, image.cols, image.rows, image_channels, d_index_image, index_image.rows, d_mosaic_image, mosaic_image.cols, d_result_image, result_image.cols, tile_size, new_tile_size, colors_per_thread, n_blocks, n_threads, tiles_per_block, tiles_per_thread, n_horizontal_tiles, n_horizontal_tiles * n_vertical_tiles);


        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            std::cout << hipGetErrorString(cudaStatus) << 0 << std::endl;
            return cudaStatus;
        }

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            std::cout << hipGetErrorString(cudaStatus) << 1 << std::endl;
            return cudaStatus;
        }

        // Calculate time
        gettimeofday(&tval_after, NULL);
        timersub(&tval_after, &tval_before, &tval_result);

        // Show results
        printf("Time elapsed: %ld.%06ld\n", (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);
        std::cout << "bloques: " << n_blocks << std::endl;
        std::cout << "hilos por bloque: " << n_threads << std::endl;

        //if (cudaStatus != hipSuccess) throw "hipDeviceSynchronize returned error after launching kernel!";

        // Copy output vector from GPU buffer to host memory.

        cudaStatus = hipMemcpy(result_image.data, d_result_image, result_image.rows * result_image.cols * result_image.channels() * sizeof(uchar), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            std::cout << hipGetErrorString(cudaStatus) << 2 << std::endl;
            return cudaStatus;
        }
        // if (cudaStatus != hipSuccess) throw "hipMemcpy failed! result image to host";
    }
    catch (char* message)
    {
        hipFree(d_image);
        hipFree(d_index_image);
        hipFree(d_mosaic_image);
        hipFree(d_result_image);
        std::cout << message << std::endl;
    }
    hipFree(d_image);
    hipFree(d_index_image);
    hipFree(d_mosaic_image);
    hipFree(d_result_image);
    return cudaStatus;
}