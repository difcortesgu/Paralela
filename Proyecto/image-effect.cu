#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>

#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/opencv.hpp>
#include <string>
#include <stdio.h>
#include <sys/time.h>

hipError_t mosaicImage(
    cv::Mat image, 
    cv::Mat index_image,
    cv::Mat mosaic_image, 
    cv::Mat result_image, 
    int tile_size, 
    int new_tile_size, 
    int n_vertical_tiles, 
    int n_horizontal_tiles, 
    int n_blocks, 
    int n_threads
);

__global__ void filterImagekernel(
    const uchar* image, 
    const uchar* index_image, 
    const uchar* mosaic_image, 
    int image_width, 
    int image_height, 
    int image_channels, 
    int blocks, 
    int threads, 
    int tiles_per_block, 
    int tiles_per_thread, 
    uchar* result_image, 
    int tile_size, 
    int new_tile_size,
    int colors_per_thread
)
{
    extern __shared__ uchar index[];

    for (int i = threadIdx.x * colors_per_thread; i < (threadIdx.x + 1) * colors_per_thread; i++)
    {
        index[i * image_channels] = index_image[i * image_channels]
        index[i * image_channels + 1] = index_image[i * image_channels + 1]
        index[i * image_channels + 2] = index_image[i * image_channels + 2]
    }    
    __syncthreads();

    // Get iteration boundaries
    int initial_tile_row = blockIdx.x * tiles_per_block
    int final_tile_row = (blockIdx.x + 1) * tiles_per_block

    int initial_tile_col = threadIdx.x * tiles_per_block
    int final_tile_col = (threadIdx.x + 1) * tiles_per_block

    double newPixelB, newPixelG, newPixelR


    // Loop rows of tiles
    for (int ti = initial_tile_row; ti < final_tile_row; ti++)
    {
        // Loop cols of tiles
        for (int tj = initial_tile_col; tj < final_tile_col; tj++)
        {
            // Start of the tile
            int pos = (ti * tile_size * image_width * channels) + (tj * tile_size * channels)

            newPixelB = 0;
            newPixelG = 0;
            newPixelR = 0;

            // Loop rows of pixels of the tile
            for (int pi = 0; pi < tile_size; pi++)
            {
                // Loop cols of pixels of the tile
                for (int pj = 0; pj < tile_size; pj++)
                {
                    // Get pixel
                    int pix_pos = pos + (pi * image_width * channels) + (pj * channels)

                    // Add the values to the new pixel
                    newPixelB += (int)image[pix_pos];
                    newPixelG += (int)image[pix_pos + 1];
                    newPixelR += (int)image[pix_pos + 2];
                }
            }

            newPixelB /= (tile_size * tile_size)
            newPixelG /= (tile_size * tile_size)
            newPixelR /= (tile_size * tile_size)

            // Get the closest color
            float minDiff = 10000;
            int closest_pos = 0;

            for (int i = 0; i < index_image_size; i += channels)
            {
                float diff = norm3d(newPixelB-index_image[i], newPixelG-index_image[i+1],newPixelR-index_image[i+2])
                if (diff < minDiff)
                {
                    minDiff = diff;
                    closest_pos = i;
                }
            }

            int pos = (ti * new_tile_size * result_image_width * channels) + (tj * new_tile_size * channels)

            // Update new image pixels with the "pixel" image
            for (int ni = 0; ni < new_tile_size; ni++)
            {
                for (int nj = 0; nj < new_tile_size; nj++)
                {
                    int pix_pos = pos + (ni * result_image_width * channels) + (nj * channels)

                    result_image[pix_pos] = index_image[((closest_pos + i) * mosaicImage_width * channels) + j * channels];
                    result_image[pix_pos + 1] = index_image[((closest_pos + i) * mosaicImage_width * channels) + j * channels + 1];
                    result_image[pix_pos + 2] = index_image[((closest_pos + i) * mosaicImage_width * channels) + j * channels + 2];
                }
            }
        }
    }
}

int main(int argc, char** argv)
{
    //Check number of arguments
    if (argc < 5)
    {
        std::cout << "Ingrese todos los argumentos necesarios para ejecutar el proceso" << std::endl;
        return -1;
    }

    int tile_size = 10, new_tile_size = 10, n_blocks = 24, n_threads = 128;

    //Get the arguments
    std::string path_image = argv[1];
    std::string path_save = argv[2];
    std::string path_index_image = "./imagenes/indexImage.jpg";
    std::string path_mosaic_image = "./imagenes/mosaicImage.jpg";

    n_blocks = atoi(argv[3]);
    if (n_blocks == 0)
    {
        std::cout << "El número de bloques es invalido" << std::endl;
        return -1;
    }

    n_threads = atoi(argv[4]);
    if (n_threads == 0)
    {
        std::cout << "El número de hilos es invalido" << std::endl;
        return -1;
    }

    tile_size = atoi(argv[4]);
    if (tile_size == 0)
    {
        cout << "El argumento de tamaño es invalido" << endl;
        return -1;
    }

    new_tile_size = atoi(argv[5]);
    if (new_tile_size == 0)
    {
        cout << "El argumento de tamaño nuevo es invalido" << endl;
        return -1;
    }

    cv::Mat image = cv::imread(path_image, cv::IMREAD_COLOR); // Read the file
    if (!image.data)
    {
        std::cout << "Could not open or find the image" << std::endl;
        return -1;
    }

    cv::Mat index_image = cv::imread(path_index_image, cv::IMREAD_COLOR); // Read the file
    if (!image.data)
    {
        std::cout << "Could not open or find the image" << std::endl;
        return -1;
    }

    cv::Mat mosaic_image = cv::imread(path_mosaic_image, cv::IMREAD_COLOR); // Read the file
    if (!image.data)
    {
        std::cout << "Could not open or find the image" << std::endl;
        return -1;
    }

    int n_vertical_tiles = std::ceil((float)image.rows / (float)tile_size);
    int n_horizontal_tiles = std::ceil((float)image.cols / (float)tile_size);

    cv::Mat result_image = cv::Mat(n_vertical_tiles * tile_size, n_horizontal_tiles * tile_size, CV_8UC3, cv::Scalar(0, 0, 0));

    // Add vectors in parallel.
    hipError_t cudaStatus = mosaicImage(image, index_image, mosaic_image, result_image, tile_size, new_tile_size, n_vertical_tiles, n_horizontal_tiles, n_blocks, n_threads);
    if (cudaStatus != hipSuccess) {
        printf(stderr, "%s\n", hipGetErrorString(cudaStatus));
        return 1;
    }

    if (!cv::imwrite(path_save, result_image)) {
        std::cout << "Could not save the image" << std::endl;
        return -1;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprint("hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

hipError_t filterImage(cv::Mat image, cv::Mat index_image, cv::Mat mosaic_image, cv::Mat result_image, int tile_size, int new_tile_size, int n_vertical_tiles, int n_horizontal_tiles, int n_blocks, int n_threads)
{
    //Declare pointers and variables
    uchar *d_image, *d_index_image, *d_mosaic_image, *d_result_image;

    int image_width = image.rows;
    int image_height = image.cols;
    int image_channels = image.channels();

    hipError_t cudaStatus = hipSuccess;

    try
    {
        // Choose which GPU to run on, change this on a multi-GPU system.
        cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess) throw "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?";

        // Allocate GPU buffers.
        cudaStatus = hipMalloc((void**)&d_image, image.rows * image.cols * image.channels() * sizeof(uchar));
        if (cudaStatus != hipSuccess) throw "hipMalloc failed! image";

        cudaStatus = hipMalloc((void**)&d_index_image, index_image.rows * index_image.cols * index_image.channels() * sizeof(uchar));
        if (cudaStatus != hipSuccess) throw "hipMalloc failed! image";

        cudaStatus = hipMalloc((void**)&d_mosaic_image, mosaic_image.rows * mosaic_image.cols * mosaic_image.channels() * sizeof(uchar));
        if (cudaStatus != hipSuccess) throw "hipMalloc failed! image";

        cudaStatus = hipMalloc((void**)&d_result_image, result_image.rows * result_image.cols * result_image.channels() * sizeof(uchar));
        if (cudaStatus != hipSuccess) throw "hipMalloc failed! result image";


        // Copy input vectors from host memory to GPU buffers.
        cudaStatus = hipMemcpy(d_image, image.data, image.rows * image.cols * image.channels() * sizeof(uchar), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) throw "hipMemcpy failed! image to device";

        cudaStatus = hipMemcpy(d_index_image, index_image.data, index_image.rows * index_image.cols * index_image.channels() * sizeof(uchar), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) throw "hipMemcpy failed! result image to device";

        cudaStatus = hipMemcpy(d_mosaic_image, mosaic_image.data, mosaic_image.rows * mosaic_image.cols * mosaic_image.channels() * sizeof(uchar), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) throw "hipMemcpy failed! result image to device";

        int tiles_per_block = std::ceil((float)n_vertical_tiles / (float)n_blocks);
        int tiles_per_thread = std::ceil((float)n_horizontal_tiles / (float)n_threads);
        int index_image_size = index_image.rows * index_image.cols * index_image.channels() * sizeof(uchar);

        // Get initial time
        struct timeval tval_before, tval_after, tval_result;
        gettimeofday(&tval_before, NULL);


        // Launch a kernel on the GPU with one thread for each element.
        filterImagekernel <<<n_blocks, n_threads, index_image_size >>> (d_image, d_index_image, d_mosaic_image, image_width, image_height, image_channels, n_blocks, n_threads, tiles_per_block, tiles_per_thread, d_result_image, tile_size, new_tile_size);


        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            std::cout << hipGetErrorString(cudaStatus) << 0 << std::endl;
            return cudaStatus;
        }

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            std::cout << hipGetErrorString(cudaStatus) << 1 << std::endl;
            return cudaStatus;
        }

        // Calculate time
        gettimeofday(&tval_after, NULL);
        timersub(&tval_after, &tval_before, &tval_result);

        // Show results
        printf("Time elapsed: %ld.%06ld\n", (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);
        std::cout<<"bloques: "<< n_blocks << std::endl;
        std::cout<<"hilos por bloque: "<< n_threads << std::endl;


        //if (cudaStatus != hipSuccess) throw "hipDeviceSynchronize returned error after launching kernel!";

        // Copy output vector from GPU buffer to host memory.

        cudaStatus = hipMemcpy(result_image.data, d_result_image, image.rows * image.cols * image.channels() * sizeof(uchar), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            std::cout << hipGetErrorString(cudaStatus) << 2 << std::endl;
            return cudaStatus;
        }
        // if (cudaStatus != hipSuccess) throw "hipMemcpy failed! result image to host";
    }
    catch (char* message)
    {
        hipFree(d_image);
        hipFree(d_result_image);
        hipFree(d_kernel);
        std::cout << message << std::endl;
    }
    hipFree(d_image);
    hipFree(d_index_image);
    hipFree(d_mosaic_image);
    hipFree(d_result_image);
    return cudaStatus;
}