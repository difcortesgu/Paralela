#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/opencv.hpp>

#include <stdio.h>

hipError_t filterImage(cv::Mat image, int *kernel, int kernel_size, cv::Mat result_image);

__global__ void filterImagekernel(const uchar* image, const int* kernel, float kernel_total, int kernel_size, int image_width, int image_height, int image_channels, int blocks, int threads, uchar* result_image)
{
    //printf("kernel_total: %f \nkernel_size: %d \nimage_width: %d \nimage_height: %d \nimage_channels: %d\n", kernel_total, kernel_size, image_width, image_height, image_channels);

    //printf("kernel:\n");
    //for (int i = 0; i < kernel_size; i++) {
    //    for (int j = 0; j < kernel_size; j++)
    //    {
    //        printf("%d ", kernel[i * kernel_size + j]);
    //    }
    //    printf("\n");
    //}

    //printf("image:\n");

    //for (int row = 0; row < image_height; row++) 
    //{
    //    for (int col = 0; col < image_width; col++)
    //    {
    //        for (int channel = 0; channel < image_channels; channel++)
    //        {
    //            printf("%d ", image[row * image_width * image_channels + col * image_channels + channel]);
    //        }
    //        printf("\t");
    //    }
    //    printf("\n");
    //}
    extern __shared__ unsigned char buffer[];

    int initial_row = blockIdx.x * (image_height / blocks);
    int final_row = blockIdx.x + 1 * (image_height / blocks);

    if (blockIdx.x == blocks - 1)
        final_row = image_height;

    int initial_col = threadIdx.x * (image_width / threads);
    int final_col = threadIdx.x + 1 * (image_width / threads);

    if (threadIdx.x == threads - 1)
        final_col = image_width;


    for (int row = initial_row ; row < final_row; row++)
    {
        for (int col = initial_col; col < final_col; col++)
        {
            for (int channel = 0; channel < image_channels; channel++)
            {
                buffer[row * image_width * image_channels + col * image_channels + channel] = image[row * image_width * image_channels + col * image_channels + channel];
            }
        }
    }

    //__syncthreads();


    initial_row = blockIdx.x * ((image_height - kernel_size + 1) / blocks);
    final_row = blockIdx.x + 1 * ((image_height - kernel_size + 1) / blocks);

    if (blockIdx.x == blocks - 1)
        final_row = (image_height - kernel_size + 1);

    initial_col = threadIdx.x * ((image_height - kernel_size + 1) / threads);
    final_col = threadIdx.x + 1 * ((image_height - kernel_size + 1) / threads);

    if (threadIdx.x == threads - 1)
        final_col = (image_height - kernel_size + 1);

    int newPixelR;
    int newPixelG;
    int newPixelB;

    for (int row = initial_row; row < final_row; row++)
    {
        for (int col = initial_col; col < final_col; col++)
        {
            newPixelR = 0;
            newPixelG = 0;
            newPixelB = 0;

            // Loop kernel rows
            for (int krow = 0; krow < kernel_size; krow++)
            {
                // Loop kernel cols
                for (int kcol = 0; kcol < kernel_size; kcol++)
                {
                    int pos = (row + krow) * image_width * image_channels + (col + kcol) * image_channels;
                    int kpos = krow * kernel_size + kcol;

                    // Update new pixel sum
                    newPixelB += (kernel[kpos] * (int)buffer[pos]);
                    newPixelG += (kernel[kpos] * (int)buffer[pos + 1]);
                    newPixelR += (kernel[kpos] * (int)buffer[pos + 2]);
                }
            }

            //  Normalize pixel and bound it
            newPixelB /= kernel_total;
            if (newPixelB < 0) newPixelB = 0;
            if (newPixelB < 255) newPixelB = 255;

            newPixelG /= kernel_total;
            if (newPixelG < 0) newPixelG = 0;
            if (newPixelG < 255) newPixelG = 255;

            newPixelR /= kernel_total;
            if (newPixelR < 0) newPixelR = 0;
            if (newPixelR < 255) newPixelR = 255;
            
            printf("R:%d G:%d B:%d\n", newPixelR, newPixelG, newPixelB);
            // Update new image pixel
            result_image[(row + (kernel_size / 2)) * image_width * image_channels + (col + (kernel_size / 2)) * image_channels] = (uchar)newPixelB;
            result_image[(row + (kernel_size / 2)) * image_width * image_channels + (col + (kernel_size / 2)) * image_channels + 1] = (uchar)newPixelG;
            result_image[(row + (kernel_size / 2)) * image_width * image_channels + (col + (kernel_size / 2)) * image_channels + 2] = (uchar)newPixelR;

        }
    }
}

int main()
{


    //NVIDIA GeForce GTX 1050 Ti
    //    CUDA Driver Version / Runtime Version          11.4 / 11.4
    //    CUDA Capability Major / Minor version number : 6.1
    //    Total amount of global memory : 4096 MBytes(4294967296 bytes)
    //    (006) Multiprocessors, (128) CUDA Cores / MP : 768 CUDA Cores
    //    Memory Bus Width : 128 - bit
    //    L2 Cache Size : 1048576 bytes
    //    Total amount of constant memory : 65536 bytes
    //    Total amount of shared memory per block : 49152 bytes
    //    Total shared memory per multiprocessor : 98304 bytes
    //    Total number of registers available per block : 65536
    //    Warp size : 32
    //    Maximum number of threads per multiprocessor : 2048
    //    Maximum number of threads per block : 1024

    cv::Mat image = cv::imread("C:\\Users\\User\\source\\repos\\test\\lenna.png", cv::IMREAD_COLOR); // Read the file
    if (!image.data)
    {
        std::cout << "Could not open or find the image" << std::endl;
        return -1;
    }
    
    cv::Mat result_image = cv::Mat(image.rows, image.cols, CV_8UC3, cv::Scalar(0, 0, 0));

    int kernel[9] = {
        0, 1, 0,
        1, -4, 1,
        0, 1, 0
    };

    // Add vectors in parallel.
    hipError_t cudaStatus = filterImage(image, kernel, 3, result_image);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Filter image failed!");
        return 1;
    }

    if (!cv::imwrite("C:\\Users\\User\\source\\repos\\test\\lenna_out.png", result_image)) {
        std::cout << "Could not save the image" << std::endl;
        return -1;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

hipError_t filterImage(cv::Mat image, int *kernel, int kernel_size, cv::Mat result_image)
{

    for (int i = 0; i < kernel_size; i++)
    {
        for (int j = 0; j < kernel_size; j++)
        {
            std::cout << kernel[i*kernel_size + j] << " ";
        }
        std::cout << std::endl;
    }

    //Declare pointers and variables
    uchar* d_image;
    uchar* d_result_image;
    int* d_kernel;
    int kernel_total = 1;
    int image_width = image.rows;
    int image_height = image.cols;
    int image_channels = image.channels();

    hipError_t cudaStatus = hipSuccess;

    try
    {
        // Choose which GPU to run on, change this on a multi-GPU system.
        cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess) throw "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?";

        // Allocate GPU buffers.
        cudaStatus = hipMalloc((void**)&d_image, image.rows * image.cols * image.channels() * sizeof(uchar));
        if (cudaStatus != hipSuccess) throw "hipMalloc failed! image";

        cudaStatus = hipMalloc((void**)&d_kernel, kernel_size * kernel_size * sizeof(int));
        if (cudaStatus != hipSuccess) throw "hipMalloc failed! kernel";

        cudaStatus = hipMalloc((void**)&d_result_image, image.rows * image.cols * image.channels() * sizeof(uchar));
        if (cudaStatus != hipSuccess) throw "hipMalloc failed! result image";


        // Copy input vectors from host memory to GPU buffers.
        cudaStatus = hipMemcpy(d_image, image.data, image.rows * image.cols * image.channels() * sizeof(uchar), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) throw "hipMemcpy failed! image to device";

        cudaStatus = hipMemcpy(d_kernel, kernel, kernel_size * kernel_size * sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) throw "hipMemcpy failed! kernel to device";

        cudaStatus = hipMemcpy(d_image, image.data, image.rows * image.cols * image.channels() * sizeof(uchar), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) throw "hipMemcpy failed! result image to device";

        int blocks = 24;
        int threads_per_block = 128;

        int shared_buffer_size = image_width * image_height * image_channels / blocks ;

        // Launch a kernel on the GPU with one thread for each element.
        filterImagekernel << <blocks, threads_per_block, shared_buffer_size>> > (d_image, d_kernel, kernel_total, kernel_size, image_width, image_height, image_channels, blocks, threads_per_block, d_result_image);

        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            std::cout << hipGetErrorString(cudaStatus) << std::endl;
            throw "Error aaaaaaaaaaaaaaaaaaaa";
        }

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        //cudaStatus = hipDeviceSynchronize();
        //if (cudaStatus != hipSuccess) throw "hipDeviceSynchronize returned error after launching kernel!";

        // Copy output vector from GPU buffer to host memory.
        cudaStatus = hipMemcpy(result_image.data, d_result_image, image.rows * image.cols * image.channels() * sizeof(uchar), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) throw "hipMemcpy failed! result image to host";


    }
    catch (char *message)
    {
        hipFree(d_image);
        hipFree(d_result_image);
        hipFree(d_kernel);
        std::cerr << message << std::endl;
    }
     return cudaStatus;
}
