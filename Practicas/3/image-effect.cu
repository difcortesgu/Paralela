#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>

#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/opencv.hpp>
#include <string>
#include <stdio.h>
#include <sys/time.h>

hipError_t filterImage(cv::Mat image, cv::Mat result_image);

int kernel[9] = {
            0, 1, 0,
            1, -4, 1,
            0, 1, 0 };
int kernel_total = 1, kernel_size = 3, n_blocks = 24, n_threads = 128;


__global__ void filterImagekernel(const uchar* image, const int* kernel, float kernel_total, int kernel_size, int image_width, int image_height, int image_channels, int blocks, int threads, int rows_per_block, int cols_per_thread, uchar* result_image)
{
    extern __shared__ uchar buffer[];

    int initial_row, final_row, initial_col, final_col;


    initial_row = blockIdx.x * rows_per_block - (kernel_size / 2);
    final_row = ((blockIdx.x + 1) * rows_per_block) + (kernel_size / 2);
    rows_per_block = rows_per_block + (kernel_size - 1);

    initial_col = threadIdx.x * cols_per_thread - (kernel_size / 2);
    final_col = ((threadIdx.x + 1) * cols_per_thread) + (kernel_size / 2);
    cols_per_thread = cols_per_thread + (kernel_size - 1);

    if (initial_row < 0)
    {
        initial_row = 0;
    }
    if (initial_col < 0)
    {
        initial_col = 0;
    }


    if (final_row > image_height)
    {
        rows_per_block = image_height - initial_row;
        final_row = image_height;
    }
    if (final_col > image_width)
    {
        cols_per_thread = image_width - initial_col;
        final_col = image_width;
    }

    //printf("block: %d, thread: %d, rows: (%d, %d), cols:(%d, %d), rows_per_block: %d, cols_per_thread: %d\n", blockIdx.x, threadIdx.x, initial_row, final_row, initial_col, final_col, rows_per_block, cols_per_thread);
    for (int row = 0; row < rows_per_block; row++)
    {
        for (int col = initial_col; col < final_col; col++)
        {
            for (int channel = 0; channel < image_channels; channel++)
            {
                buffer[(row * image_width * image_channels) + (col * image_channels) + channel] = image[((initial_row + row) * image_width * image_channels) + (col * image_channels) + channel];
            }
        }
    }

    __syncthreads();

    int newPixelR, newPixelG, newPixelB;

    rows_per_block = rows_per_block - (kernel_size - 1);
    final_row = final_row - (kernel_size - 1);

    cols_per_thread = cols_per_thread - (kernel_size - 1);
    final_col = final_col - (kernel_size - 1);

    for (int row = 0; row < rows_per_block; row++)
    {
        for (int col = initial_col; col < final_col; col++)
        {
            newPixelR = 0;
            newPixelG = 0;
            newPixelB = 0;

            // Loop kernel rows
            for (int krow = 0; krow < kernel_size; krow++)
            {
                // Loop kernel cols
                for (int kcol = 0; kcol < kernel_size; kcol++)
                {
                    int pos = ((row + krow) * image_width * image_channels) + ((col + kcol) * image_channels);
                    int kpos = (krow * kernel_size) + kcol;

                    //printf("pos: %d\n", pos);

                    //// Update new pixel sum
                    newPixelB += (kernel[kpos] * (int)buffer[pos]);
                    newPixelG += (kernel[kpos + 1] * (int)buffer[pos + 1]);
                    newPixelR += (kernel[kpos + 2] * (int)buffer[pos + 2]);
                }
            }

            ////  Normalize pixel and bound it
            newPixelB /= kernel_total;
            if (newPixelB < 0) newPixelB = 0;
            if (newPixelB > 255) newPixelB = 255;

            newPixelG /= kernel_total;
            if (newPixelG < 0) newPixelG = 0;
            if (newPixelG > 255) newPixelG = 255;

            newPixelR /= kernel_total;
            if (newPixelR < 0) newPixelR = 0;
            if (newPixelR > 255) newPixelR = 255;

            //// Update new image pixel
            result_image[(initial_row + row + (kernel_size / 2)) * image_width * image_channels + (col + (kernel_size / 2)) * image_channels] = (uchar)newPixelB;
            result_image[(initial_row + row + (kernel_size / 2)) * image_width * image_channels + (col + (kernel_size / 2)) * image_channels + 1] = (uchar)newPixelG;
            result_image[(initial_row + row + (kernel_size / 2)) * image_width * image_channels + (col + (kernel_size / 2)) * image_channels + 2] = (uchar)newPixelR;

        }
    }

}

int main(int argc, char** argv)
{
    //Check number of arguments
    if (argc < 5)
    {
        std::cout << "Ingrese todos los argumentos necesarios para ejecutar el proceso" << std::endl;
        return -1;
    }

    //Get the arguments
    std::string path_image = argv[1];
    std::string path_save = argv[2];

    n_blocks = atoi(argv[3]);
    if (n_blocks == 0)
    {
        std::cout << "El número de bloques es invalido" << std::endl;
        return -1;
    }

    n_threads = atoi(argv[4]);
    if (n_threads == 0)
    {
        std::cout << "El número de hilos es invalido" << std::endl;
        return -1;
    }

    cv::Mat image = cv::imread(path_image, cv::IMREAD_COLOR); // Read the file
    if (!image.data)
    {
        std::cout << "Could not open or find the image" << std::endl;
        return -1;
    }

    cv::Mat result_image = cv::Mat(image.rows, image.cols, CV_8UC3, cv::Scalar(0, 0, 0));

    // Add vectors in parallel.
    hipError_t cudaStatus = filterImage(image, result_image);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "%s\n", hipGetErrorString(cudaStatus));
        fprintf(stderr, "Filter image failed!");
        return 1;
    }

    if (!cv::imwrite(path_save, result_image)) {
        std::cout << "Could not save the image" << std::endl;
        return -1;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

hipError_t filterImage(cv::Mat image, cv::Mat result_image)
{
    //Declare pointers and variables
    uchar* d_image;
    uchar* d_result_image;
    int* d_kernel;
    int image_width = image.rows;
    int image_height = image.cols;
    int image_channels = image.channels();

    hipError_t cudaStatus = hipSuccess;

    try
    {
        // Choose which GPU to run on, change this on a multi-GPU system.
        cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess) throw "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?";

        // Allocate GPU buffers.
        cudaStatus = hipMalloc((void**)&d_image, image.rows * image.cols * image.channels() * sizeof(uchar));
        if (cudaStatus != hipSuccess) throw "hipMalloc failed! image";

        cudaStatus = hipMalloc((void**)&d_kernel, kernel_size * kernel_size * sizeof(int));
        if (cudaStatus != hipSuccess) throw "hipMalloc failed! kernel";

        cudaStatus = hipMalloc((void**)&d_result_image, image.rows * image.cols * image.channels() * sizeof(uchar));
        if (cudaStatus != hipSuccess) throw "hipMalloc failed! result image";


        // Copy input vectors from host memory to GPU buffers.
        cudaStatus = hipMemcpy(d_image, image.data, image.rows * image.cols * image.channels() * sizeof(uchar), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) throw "hipMemcpy failed! image to device";

        cudaStatus = hipMemcpy(d_kernel, kernel, kernel_size * kernel_size * sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) throw "hipMemcpy failed! kernel to device";

        cudaStatus = hipMemcpy(d_image, image.data, image.rows * image.cols * image.channels() * sizeof(uchar), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) throw "hipMemcpy failed! result image to device";


        int rows_per_block = std::ceil((float)image_height / (float)n_blocks);
        int cols_per_thread = std::ceil((float)image_width / (float)n_threads);
        int shared_buffer_size = (rows_per_block + (kernel_size - 1)) * image_width * image_channels * sizeof(uchar);

        // Get initial time
        struct timeval tval_before, tval_after, tval_result;
        gettimeofday(&tval_before, NULL);


        // Launch a kernel on the GPU with one thread for each element.
        filterImagekernel << <n_blocks, n_threads, shared_buffer_size >> > (d_image, d_kernel, kernel_total, kernel_size, image_width, image_height, image_channels, n_blocks, n_threads, rows_per_block, cols_per_thread, d_result_image);


        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            std::cout << hipGetErrorString(cudaStatus) << 0 << std::endl;
            return cudaStatus;
        }

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            std::cout << hipGetErrorString(cudaStatus) << 1 << std::endl;
            return cudaStatus;
        }

        // Calculate time
        gettimeofday(&tval_after, NULL);
        timersub(&tval_after, &tval_before, &tval_result);

        // Show results
        printf("Time elapsed: %ld.%06ld\n", (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);
        std::cout<<"bloques: "<< n_blocks << std::endl;
        std::cout<<"hilos por bloque: "<< n_threads << std::endl;


        //if (cudaStatus != hipSuccess) throw "hipDeviceSynchronize returned error after launching kernel!";

        // Copy output vector from GPU buffer to host memory.

        cudaStatus = hipMemcpy(result_image.data, d_result_image, image.rows * image.cols * image.channels() * sizeof(uchar), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            std::cout << hipGetErrorString(cudaStatus) << 2 << std::endl;
            return cudaStatus;
        }
        // if (cudaStatus != hipSuccess) throw "hipMemcpy failed! result image to host";


    }
    catch (char* message)
    {
        hipFree(d_image);
        hipFree(d_result_image);
        hipFree(d_kernel);
        std::cout << message << std::endl;
    }
    return cudaStatus;
}
