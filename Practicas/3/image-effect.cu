#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>

#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/opencv.hpp>
#include <string>
#include <stdio.h>

hipError_t filterImage(cv::Mat image, cv::Mat result_image);
void get_kernel_info(int filter);

int kernel[9];
int kernel_total, kernel_size, n_blocks, n_threads;


__global__ void filterImagekernel(const uchar* image, const int* kernel, float kernel_total, int kernel_size, int image_width, int image_height, int image_channels, int blocks, int threads, int rows_per_block, int cols_per_thread, uchar* result_image)
{
    extern __shared__ uchar buffer[];

    int initial_row, final_row, initial_col, final_col;


    initial_row = blockIdx.x * rows_per_block - (kernel_size / 2);
    final_row = ((blockIdx.x + 1) * rows_per_block) + (kernel_size / 2);
    rows_per_block = rows_per_block + (kernel_size - 1);

    initial_col = threadIdx.x * cols_per_thread - (kernel_size / 2);
    final_col = ((threadIdx.x + 1) * cols_per_thread) + (kernel_size / 2);
    cols_per_thread = cols_per_thread + (kernel_size - 1);

    if (initial_row < 0)
    {
        initial_row = 0;
    }
    if (initial_col < 0)
    {
        initial_col = 0;
    }


    if (final_row > image_height)
    {
        rows_per_block = image_height - initial_row;
        final_row = image_height;
    }
    if (final_col > image_width)
    {
        cols_per_thread = image_width - initial_col;
        final_col = image_width;
    }

    //printf("block: %d, thread: %d, rows: (%d, %d), cols:(%d, %d), rows_per_block: %d, cols_per_thread: %d\n", blockIdx.x, threadIdx.x, initial_row, final_row, initial_col, final_col, rows_per_block, cols_per_thread);
    for (int row = 0; row < rows_per_block; row++)
    {
        for (int col = initial_col; col < final_col; col++)
        {
            for (int channel = 0; channel < image_channels; channel++)
            {
                buffer[(row * image_width * image_channels) + (col * image_channels) + channel] = image[((initial_row + row) * image_width * image_channels) + (col * image_channels) + channel];
            }
        }
    }

    __syncthreads();

    int newPixelR, newPixelG, newPixelB;

    rows_per_block = rows_per_block - (kernel_size - 1);
    final_row = final_row - (kernel_size - 1);

    cols_per_thread = cols_per_thread - (kernel_size - 1);
    final_col = final_col - (kernel_size - 1);

    for (int row = 0; row < rows_per_block; row++)
    {
        for (int col = initial_col; col < final_col; col++)
        {
            newPixelR = 0;
            newPixelG = 0;
            newPixelB = 0;

            // Loop kernel rows
            for (int krow = 0; krow < kernel_size; krow++)
            {
                // Loop kernel cols
                for (int kcol = 0; kcol < kernel_size; kcol++)
                {
                    int pos = ((row + krow) * image_width * image_channels) + ((col + kcol) * image_channels);
                    int kpos = (krow * kernel_size) + kcol;

                    //printf("pos: %d\n", pos);

                    //// Update new pixel sum
                    newPixelB += (kernel[kpos] * (int)buffer[pos]);
                    newPixelG += (kernel[kpos + 1] * (int)buffer[pos + 1]);
                    newPixelR += (kernel[kpos + 2] * (int)buffer[pos + 2]);
                }
            }

            ////  Normalize pixel and bound it
            newPixelB /= kernel_total;
            if (newPixelB < 0) newPixelB = 0;
            if (newPixelB > 255) newPixelB = 255;

            newPixelG /= kernel_total;
            if (newPixelG < 0) newPixelG = 0;
            if (newPixelG > 255) newPixelG = 255;

            newPixelR /= kernel_total;
            if (newPixelR < 0) newPixelR = 0;
            if (newPixelR > 255) newPixelR = 255;

            //// Update new image pixel
            result_image[(initial_row + row + (kernel_size / 2)) * image_width * image_channels + (col + (kernel_size / 2)) * image_channels] = (uchar)newPixelB;
            result_image[(initial_row + row + (kernel_size / 2)) * image_width * image_channels + (col + (kernel_size / 2)) * image_channels + 1] = (uchar)newPixelG;
            result_image[(initial_row + row + (kernel_size / 2)) * image_width * image_channels + (col + (kernel_size / 2)) * image_channels + 2] = (uchar)newPixelR;

        }
    }

}

int main(int argc, char** argv)
{
    // Check number of arguments
    if (argc < 5)
    {
        std::cout << "Ingrese todos los argumentos necesarios para ejecutar el proceso" << std::endl;
        return -1;
    }

    // Get the arguments
    std::string path_image = argv[1];
    std::string path_save = argv[2];

    n_blocks = atoi(argv[3]);
    if (n_blocks == 0)
    {
        std::cout << "El número de bloques es invalido" << std::endl;
        return -1;
    }

    n_threads = atoi(argv[4]);
    if (n_threads == 0)
    {
        std::cout << "El número de hilos es invalido" << std::endl;
        return -1;
    }
    int filter = 0;
    if (argc > 5)
    {
        filter = atoi(argv[5]);
        if (filter == 0)
        {
            std::cout << "El numero del filtro es invalido" << std::endl;
            return -1;
        }
    }

    cv::Mat image = cv::imread(path_image, cv::IMREAD_COLOR); // Read the file
    if (!image.data)
    {
        std::cout << "Could not open or find the image" << std::endl;
        return -1;
    }

    cv::Mat result_image = cv::Mat(image.rows, image.cols, CV_8UC3, cv::Scalar(0, 0, 0));

    // Add vectors in parallel.
    hipError_t cudaStatus = filterImage(image, result_image);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "%s\n", hipGetErrorString(cudaStatus));
        fprintf(stderr, "Filter image failed!");
        return 1;
    }

    if (!cv::imwrite(path_save, result_image)) {
        std::cout << "Could not save the image" << std::endl;
        return -1;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

hipError_t filterImage(cv::Mat image, cv::Mat result_image)
{
    //Declare pointers and variables
    uchar* d_image;
    uchar* d_result_image;
    int* d_kernel;
    int image_width = image.rows;
    int image_height = image.cols;
    int image_channels = image.channels();

    hipError_t cudaStatus = hipSuccess;

    try
    {
        // Choose which GPU to run on, change this on a multi-GPU system.
        cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess) throw "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?";

        // Allocate GPU buffers.
        cudaStatus = hipMalloc((void**)&d_image, image.rows * image.cols * image.channels() * sizeof(uchar));
        if (cudaStatus != hipSuccess) throw "hipMalloc failed! image";

        cudaStatus = hipMalloc((void**)&d_kernel, kernel_size * kernel_size * sizeof(int));
        if (cudaStatus != hipSuccess) throw "hipMalloc failed! kernel";

        cudaStatus = hipMalloc((void**)&d_result_image, image.rows * image.cols * image.channels() * sizeof(uchar));
        if (cudaStatus != hipSuccess) throw "hipMalloc failed! result image";


        // Copy input vectors from host memory to GPU buffers.
        cudaStatus = hipMemcpy(d_image, image.data, image.rows * image.cols * image.channels() * sizeof(uchar), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) throw "hipMemcpy failed! image to device";

        cudaStatus = hipMemcpy(d_kernel, kernel, kernel_size * kernel_size * sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) throw "hipMemcpy failed! kernel to device";

        cudaStatus = hipMemcpy(d_image, image.data, image.rows * image.cols * image.channels() * sizeof(uchar), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) throw "hipMemcpy failed! result image to device";


        int rows_per_block = std::ceil((float)image_height / (float)n_blocks);
        int cols_per_thread = std::ceil((float)image_width / (float)n_threads);
        int shared_buffer_size = (rows_per_block + (kernel_size - 1)) * image_width * image_channels * sizeof(uchar);

        // Launch a kernel on the GPU with one thread for each element.
        filterImagekernel << <n_blocks, n_threads, shared_buffer_size >> > (d_image, d_kernel, kernel_total, kernel_size, image_width, image_height, image_channels, n_blocks, n_threads, rows_per_block, cols_per_thread, d_result_image);

        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            std::cout << hipGetErrorString(cudaStatus) << 0 << std::endl;
            return cudaStatus;
        }

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            std::cout << hipGetErrorString(cudaStatus) << 1 << std::endl;
            return cudaStatus;
        }

        //if (cudaStatus != hipSuccess) throw "hipDeviceSynchronize returned error after launching kernel!";

        // Copy output vector from GPU buffer to host memory.

        cudaStatus = hipMemcpy(result_image.data, d_result_image, image.rows * image.cols * image.channels() * sizeof(uchar), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            std::cout << hipGetErrorString(cudaStatus) << 2 << std::endl;
            return cudaStatus;
        }
        // if (cudaStatus != hipSuccess) throw "hipMemcpy failed! result image to host";


    }
    catch (char* message)
    {
        hipFree(d_image);
        hipFree(d_result_image);
        hipFree(d_kernel);
        std::cout << message << std::endl;
    }
    return cudaStatus;
}


void get_kernel_info(int filter)
{
    // Get kernel sum to use it later
    std::vector<int> temp_kernel;
 
    bool brillo = false;
    switch (filter)
    {
    case 1:
        //DETECCION DE BORDES
        temp_kernel = {
            0, 1, 0,
            1, -4, 1,
            0, 1, 0 };
        break; //optional
    case 2:
        // REPUJADO
        temp_kernel = {
            -2, -1, 0,
            -1, 1, 1,
            0, 1, 2 };

        break; //optional
    case 3:
        // DESENFOCADO 3x3
        temp_kernel = {
            1, 1, 1,
            1, 1, 1,
            1, 1, 1 };

        break; //optional
    case 4:
        // ENFOCADO
        temp_kernel = {
            0, -1, 0,
            -1, 5, -1,
            0, -1, 0 };
        kernel_size = 3;
        break; //optional
    case 5:
        // brillo bajo
        temp_kernel = {
            0, 0, 0,
            0, 1, 0,
            0, 0, 0 };
        brillo = true;
        kernel_total = 1.5;
        break;
    case 6:
        // brillo alto
        temp_kernel = {
            0, 0, 0,
            0, 1, 0,
            0, 0, 0 };
        brillo = true;
        kernel_total = 0.5;
        break;
    default:   //Identidad
        temp_kernel = {
            0, 0, 0,
            0, 1, 0,
            0, 0, 0 };
    }
    if (!brillo)
    {
        for (int i = 0; i < 9; i++)
        {
            kernel_total += temp_kernel[i];
            kernel[i] = temp_kernel[i];
        }

        if (kernel_total == 0)
            kernel_total = 1;
    }

}
